#include "hip/hip_runtime.h"

#include "common.cpp"
#include "svm_data.h"
#include <hip/hip_runtime_api.h>
#include <hipblas.h>
#include ""
__global__ void dot_prod_dense(float *X, float *Z, int nrows, int ncols)
{
	unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
	const unsigned int shift = gridDim.x*blockDim.x;
	if (i < nrows)
	{
		float buf = 0;

		for (int j = 0; j < ncols; ++j)
		{			
			buf +=X[i*ncols+j]*X[i*ncols+j];
		}
		Z[i] = buf;

	}
	__syncthreads();
}
// C = X * Y[i] : i = 0..nrows-1
__global__ void dot_line(float *X, float *Y, float *Z, int nrows, int ncols)
{
	unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
	const unsigned int shift = gridDim.x*blockDim.x;
	if (i < nrows)
	{
		float buf = 0;

		for (int j = 0; j < ncols; ++j)
		{			
			buf +=X[j]*Y[i*ncols+j];
		}
		Z[i] = buf;

	}
	__syncthreads();
}


__global__ void reduction( float* d_k, float *d_dotSV, float *d_dotTV, float *d_koef, int nSV, int irow, int offset, float gamma, int kernelcode, float *result)
{
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int shift = gridDim.x*blockDim.x;
	const unsigned int blockdim = blockDim.x;
	const unsigned int tid = threadIdx.x;
	__shared__ float reduction [MAXTHREADS];
	if (i < nSV)
	{
		if(kernelcode == 0)	
		{
			float val =  gamma * (2*d_k[irow*nSV+i]-d_dotSV[i]-d_dotTV[irow+offset]);
			reduction[tid] =  d_koef[i]*expf(val);
		}
	}
	__syncthreads();

	if(blockdim>=512)	{if(tid<256){reduction[tid] += reduction[tid + 256];}__syncthreads();}
	if(blockdim>=256)	{if(tid<128){reduction[tid] += reduction[tid + 128];}__syncthreads();}
	if(blockdim>=128)   {if(tid<64)	{reduction[tid] += reduction[tid + 64];}__syncthreads();}
	if(tid<32){	if(blockdim >= 64)	{reduction[tid] += reduction[tid + 32];}
	if(blockdim >= 32)	{reduction[tid] += reduction[tid + 16];}
	if(blockdim >= 16)	{reduction[tid] += reduction[tid + 8];}
	if(blockdim >= 8)	{reduction[tid] += reduction[tid + 4];}
	if(blockdim >= 4)	{reduction[tid] += reduction[tid + 2];}
	if(blockdim >= 2)	{reduction[tid] += reduction[tid + 1];}	}

	if(tid==0){	result[blockIdx.x]=reduction[tid];}
}

void classifier(svm_model *model, svm_test *test, int *h_l_estimated )
{
	int nTV = test->nTV;
	int nSV = model->nSV;
	int nfeatures = model->nfeatures;

	float *d_TV = 0;
	

	float *d_SV = 0;
	hipMalloc((void**) &d_SV, nSV*nfeatures*sizeof(float));
	hipMemcpy(d_SV, model->SV_dens, nSV*nfeatures*sizeof(float),hipMemcpyHostToDevice);

	float *d_l_SV = 0;
	hipMalloc((void**) &d_l_SV, nSV*sizeof(float));
	hipMemcpy(d_l_SV, model->l_SV, nSV*sizeof(float),hipMemcpyHostToDevice);

	float *d_dotTV = 0;
	hipMalloc((void**) &d_dotTV, nTV*sizeof(float)); 

	float *d_dotSV = 0;
	hipMalloc((void**) &d_dotSV, nSV*sizeof(float)); 

	// Allocate device memory for F
	


	void* temp;
	size_t pitch;
	hipMallocPitch(&temp, &pitch, nSV * sizeof(float),1);
	hipFree(temp);
	unsigned int remainingMemory = 0;
	unsigned int totalMemory = 0;
	hipMemGetInfo(&remainingMemory, &totalMemory);	
	int cache_size = remainingMemory/pitch; // # of TVs in cache
	if (nTV <= cache_size)
	{
		cache_size = nTV;
	}
	float *d_k = 0;
	hipMalloc((void**)&d_k, cache_size*nSV * sizeof(float));
	hipMalloc((void**) &d_TV, cache_size*nfeatures*sizeof(float));
	int nthreads = MAXTHREADS;
	int nblocksSV = min(MAXBLOCKS, (nSV + nthreads - 1)/nthreads);
	int nblocksTV = min(MAXBLOCKS, (nTV + nthreads - 1)/nthreads);
	int nblocks_cache = min(MAXBLOCKS, (cache_size + nthreads - 1)/nthreads);

	float* h_fdata= (float*) malloc(nblocks_cache*sizeof(float));
	float* d_fdata=0;
	hipMalloc((void**) &d_fdata, nblocks_cache*sizeof(float));
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_DEVICE);

	/*float* dot= (float*) malloc(nSV*nTV*sizeof(float));
	float* dot1= (float*) malloc(nSV*nTV*sizeof(float));
	float* dot2= (float*) malloc(nSV*sizeof(float));
	for(int i= 0; i<nSV; i++)
		hipblasSdot(handle, nfeatures, &d_SV[i*nfeatures], 1, &d_SV[i*nfeatures], 1, &d_dotSV[i]);
	for(int i= 0; i<nTV; i++)
		hipblasSdot(handle, nfeatures, &d_TV[i*nfeatures], 1, &d_TV[i*nfeatures], 1, &d_dotTV[i]);	*/
	dot_prod_dense<<<nblocksSV, nthreads>>>(d_SV, d_dotSV, nSV, nfeatures);
	dot_prod_dense<<<nblocksTV, nthreads>>>(d_TV, d_dotTV, nTV, nfeatures);
	int cc=0;
	int offset = 0;
	float alfa = 1.;
	float betta = 0;
	int num_of_parts =  (nTV + cache_size - 1)/cache_size;
	for (int ipart = 0; ipart < num_of_parts; ipart++)
	{
		if ((ipart == (num_of_parts - 1)) && ((nTV - offset) != 0) )
		{
			cache_size = nTV - offset;
		}
			//Allocate Kernel Cache Memory
		hipMemcpy(&d_TV[offset*nfeatures], test->TV, cache_size*nfeatures*sizeof(float),hipMemcpyHostToDevice);
		for (int i = 0; i < cache_size; i++)
		{
			dot_line<<<nblocks_cache, nthreads>>>(&d_TV[i*nfeatures], d_SV, &d_k[i*nSV], cache_size, nfeatures);
			//hipMemcpy(dot, &d_k[i*nSV], nSV*sizeof(float), hipMemcpyDeviceToHost);
			/*for (int l = 0; l < nSV; l++)
			{
				hipblasSdot(handle, nfeatures, &d_TV[i*nfeatures], 1, &d_SV[l*nfeatures], 1, &d_k[i*nSV+l]);
			}*/

			reduction<<<nblocks_cache, nthreads>>>(d_k, d_dotSV, d_dotTV, d_l_SV, nSV, i, offset, model->coef_gamma, model->kernel_type, d_fdata);
			hipMemcpy(h_fdata, d_fdata, nblocks_cache*sizeof(float), hipMemcpyDeviceToHost);
			float sum = 0;
			for (int k = 0; k < nblocks_cache; k++)
			{
				sum += h_fdata[k];
			}
			sum += model->b[0];
			if (sum > 0)
			{
				h_l_estimated[i + offset] = model->label_set[0];
			}
			else
			{
				h_l_estimated[i + offset] = model->label_set[1];
			}
		}

		offset += cache_size;
	}
	hipblasDestroy(handle);
}
int main(int argc, char **argv)
{
	FILE *input;
	argc = 4;
	argv[1] = "C:\\Data\\b.txt";
	argv[2] = "C:\\Data\\b.model";
	argv[3] = "10";

	if(argc<4)
		exit_with_help();
	struct svm_model *model = (svm_model*)malloc(sizeof(svm_model));
	struct svm_test *test = (svm_test*)malloc(sizeof(svm_test));
	sscanf(argv[3],"%d",&model->nfeatures);

	if(read_model(argv[2], model, model->nfeatures)==0)
	{
		fprintf(stderr,"can't read model %s\n",argv[2]);
		exit(1);
	}

	if((input = fopen(argv[1],"r")) == NULL)
	{
		fprintf(stderr,"can't open input file %s\n",argv[1]);
		exit(1);
	}
	parse_TV(input,&test->TV,&test->l_TV,&test->nTV,model->nfeatures);
	fclose(input);

	int* h_estimated_labels = (int*)malloc(model->nSV*sizeof(int));
	
	classifier(model, test, h_estimated_labels);

	int errors=0;

	for (int i=0; i<test->nTV; i++)
	{
		if( test->l_TV[i]!=h_estimated_labels[i])
		{
			errors++;
		}
	}
	printf("# of testing samples %d, # errors %d, Rate %f\n", test->nTV, errors, 100* (float) (test->nTV -errors)/(float)test->nTV);

	return 0;
}
